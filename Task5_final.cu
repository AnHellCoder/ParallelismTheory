#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <mpi.h>

using namespace std;

__global__ void compute(double* arrnew, double* arrprev, int size, int groupSize){
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < groupSize - 1 && j > 0 && j < size - 1) {
        arrnew[i * size + j] = 0.25 * (arrprev[i * size + j - 1] + arrprev[(i - 1) * size + j] + arrprev[(i + 1) * size + j] + arrprev[i * size + j + 1]);
    }
}


__global__ void loss_calculation(double* arrnew, double* arrprev, double* arrloss){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    arrloss[i] = abs(arrprev[i] - arrnew[i]);
}

int main(int argc, char** argv) {
    clock_t begin = clock();

    int rank, total_ranks;
    MPI_Init(&argc, &argv);

    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &total_ranks);


    hipSetDevice(rank);

    if (rank!=0)
        hipDeviceEnablePeerAccess(rank - 1, 0);
    if (rank!=total_ranks-1)
        hipDeviceEnablePeerAccess(rank + 1, 0);

    int size = 1024, lim = 1000000, iter_count = 0;
    double acc = 0.000001, loss = 1.0;

    if(argc > 1){
            if(string(argv[1]) == "--accuracy") acc = atof(argv[2]);
            else{
                    cout << "Invalid argument sent. Send '--help' argument to get help" << endl;
                    exit(0);
            }

            if(string(argv[3]) == "--size") size = atoi(argv[4]);
            else{
                    cout << "Invalid argument sent. Send '--help' argument to get help" << endl;
                    exit(0);
            }

            if(string(argv[5]) == "--limit") lim = atoi(argv[6]);
            else{
                    cout << "Invalid argument sent. Send '--help' argument to get help" << endl;
                    exit(0);
            }
    }

    size_t size_per_gpu = size / total_ranks;
    size_t startRow = size_per_gpu * rank;

    double* arr;

    arr = new double[size * size];
    memset(arr, 0, size * size * sizeof(double));

    int k = size - 1;
    arr[0] = 10;
    arr[k * size] = 20;
    arr[k] = 20;
    arr[k * size + k] = 30;

    for(int i = 0; i < size; i++){
            for(int j = 0; j < size; j++) cout << arr[i * size + j] << " ";
            cout << endl;
    }

    double step = (double)10/(size-1);
    for (size_t i = 1; i < size - 1; i++) {
        arr[i] = arr[i-1] + step;
        arr[i * size] = arr[(i - 1) * size] + step;
        arr[k * size + i] = arr[k * size + (i - 1)] + step;
        arr[i * size + k] = arr[(i - 1) * size + k] + step;
    }

    for(int i = 0; i < size; i++){
            for(int j = 0; j < size; j++) cout << arr[i * size + j] << " ";
            cout << endl;
    }
    return;

    if (rank != 0 && rank != total_ranks - 1) size_per_gpu += 2;
    else size_per_gpu++;

    double* arrnew, *arrprev, *arrloss, *cudaLoss, *temp_storage = NULL;
    hipMalloc(&arrprev, sizeof(double) * (size * size_per_gpu));
    hipMalloc(&arrnew, sizeof(double) * (size * size_per_gpu));
    hipMalloc(&arrloss, sizeof(double) * (size * size_per_gpu));
    hipMalloc(&cudaLoss, sizeof(double));

    size_t temp_storage_bytes = 0;
    size_t offset = (rank != 0) ? size : 0;

    int threads_x = (size < 1024) ? size : 1024;
    int blocks_y = size_per_gpu;
    int blocks_x = size / threads_x;

    dim3 threads(threads_x, 1);
    dim3 blocks(blocks_x, blocks_y);

    hipMemcpy(arrprev, arr + (startRow * size) - offset, sizeof(double) * (size * size_per_gpu), hipMemcpyHostToDevice);
    hipMemcpy(arrnew, arr + (startRow * size) - offset, sizeof(double) * (size * size_per_gpu), hipMemcpyHostToDevice);

    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, arrloss, cudaLoss, (size * size_per_gpu));
    hipMalloc(&temp_storage, temp_storage_bytes);

    hipStream_t stream;
    hipStreamCreate(&stream);

    while (iter_count < lim && loss > acc) {
        iter_count += 1;

        compute<<<blocks, threads, 0, stream>>>(arrnew, arrprev, size, size_per_gpu);

        if(iter_count % 100 == 0){
            loss_calculation<<<blocks_x * blocks_y, threads_x, 0, stream>>>(arrnew, arrprev, arrloss);
            hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, arrloss, cudaLoss, (size * size_per_gpu));

            hipStreamSynchronize(stream);

            MPI_Allreduce(cudaLoss,cudaLoss, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

            hipMemcpyAsync(&loss, cudaLoss, sizeof(double), hipMemcpyDeviceToHost, stream);

            clock_t mid = clock();
            cout << "On " << iter_count << " iteration loss equals " << loss << endl;
            cout << "Time elapsed: " << (double)(mid - begin)/CLOCKS_PER_SEC << endl;

        }
        hipStreamSynchronize(stream);

        if (rank != 0){
            MPI_Sendrecv(arrnew + size + 1, size - 2, MPI_DOUBLE, rank - 1, 0, arrnew + 1, size - 2, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }

        if (rank != total_ranks - 1){
            MPI_Sendrecv(arrnew + (size_per_gpu - 2) * size + 1, size - 2, MPI_DOUBLE, rank + 1, 0, arrnew + (size_per_gpu - 1) * size + 1, size - 2, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }

        swap(arrprev, arrnew);
    }

    if (rank == 0){
        clock_t end = clock();

        cout << "On " << iter_count << " iteration loss descended to " << loss << endl;
        cout << "Time elapsed: " << (double)(end - begin)/CLOCKS_PER_SEC << endl;
    }

    hipFree(arrprev);
    hipFree(arrnew);
    hipFree(temp_storage);
    hipFree(cudaLoss);
    delete[] arr;

    MPI_Finalize();

    return 0;
}
