#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>


using namespace std;

void print_error(){
	cout << "Arguments were not parsed correctly!" << endl;
	cout << "Print --help to get help" << endl;
}

void print_help(){
	cout << "How to send args through cmd:" << endl;
	cout << "--accuracy <double> --size <int> --limit <int>" << endl;
}

__global__ void init(double* arr, int size){
	int k = size - 1;
	double step = (double)10/size;

	arr[0] = 10;
	arr[k] = 20;
	arr[k * size] = 20;
	arr[k * size + k] = 30;
	for(int i = 1; i < k; i++){
		arr[i] = arr[i - 1] + step;
		arr[k * size + i] = arr[k * size + (i - 1)] + step;
		arr[i * size] = arr[(i - 1) * size] + step;
		arr[i * size + k] = arr[(i - 1) * size + k] + step;
	}
}

__global__ void compute(double* arrnew, double* arrprev, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(blockIdx.x == 0 || blockIdx.x == size - 1) return;
	if(threadIdx.x == 0 || threadIdx.x == size - 1) return;

	arrnew[i] = 0.25 * (arrprev[i - 1] + arrprev[i + 1] + arrprev[i - size] + arrprev[i + size]);
}

__global__ void loss_calculate(double* arrnew, double* arrprev, double* arrloss){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	arrloss[i] = arrnew[i] - arrprev[i];
}

__global__ void printArr(double* arr, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%lf ", arr[i * size + j]);
		}
		printf("\n");
	}
}

int main(int argc, char* argv[]){
	clock_t begin = clock();
	hipSetDevice(3);

	double acc, loss = 1.0;
	int iter = 0, lim, size;

	//Argument parsing
	string* args = new string[argc];
	for(int i = 0; i < argc; i++) args[i] = argv[i];

	if(argc == 2 && args[1] == "--help"){
		print_help();
		exit(0);
	}

	if(args[1] == "--accuracy") acc = atof(argv[2]);
	else{
		print_error();
		exit(0);
	}

	if(args[3] == "--size") size = atoi(argv[4]);
	else{
		print_error();
		exit(0);
	}

	if(args[5] == "--limit") lim = atoi(argv[6]);
	else{
		print_error();
		exit(0);
	}
	//End argument parsing

	hipStream_t stream;
	hipStreamCreate(&stream);

	double* arrprev;
	double* arrnew;
	double* arrloss;
	double* temp_storage = NULL;
	double* cudaLoss;

	size_t ts_bytes;

	hipMalloc(&cudaLoss, sizeof(double));

	hipMalloc(&arrprev, sizeof(double) * (size * size));
	hipMalloc(&arrnew, sizeof(double) * (size * size));
	hipMalloc(&arrloss, sizeof(double) * (size * size));

	init<<<1, 1>>>(arrprev, size);
	init<<<1, 1>>>(arrnew, size);

	while(loss > acc && iter <= lim){
		iter++;

		compute<<<size, size>>>(arrnew, arrprev, size);

		if(iter % 100 == 0){
			loss_calculate<<<size, size>>>(arrnew, arrprev, arrloss);

			hipMalloc(&cudaLoss, sizeof(double));
			hipcub::DeviceReduce::Max(temp_storage, ts_bytes, arrloss, cudaLoss, (size * size));
			hipMalloc(&temp_storage, ts_bytes);
			hipcub::DeviceReduce::Max(temp_storage, ts_bytes, arrloss, cudaLoss, (size * size));

			hipMemcpy(&loss, cudaLoss, sizeof(double), hipMemcpyDeviceToHost);

			clock_t mid = clock();
			double te = (double)(mid - begin)/CLOCKS_PER_SEC;

			cout << "On " << iter << " iteration loss equals: " << loss << endl;
			cout << "Time elapsed: " << te << endl;
		}

		swap(arrprev, arrnew);
		//hipMemcpy(&loss, cudaLoss, sizeof(double), hipMemcpyDeviceToHost);
	}

	return 0;
}
